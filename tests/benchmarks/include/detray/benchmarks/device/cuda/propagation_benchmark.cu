#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2022-2024 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "detray/benchmarks/device/cuda/propagation_benchmark.hpp"
#include "detray/core/detector_metadata.hpp"
#include "detray/definitions/detail/cuda_definitions.hpp"
#include "detray/detectors/toy_metadata.hpp"

namespace detray {

template <typename propagator_t>
__global__ void __launch_bounds__(256, 4) propagator_benchmark_kernel(
    propagation::config cfg,
    typename propagator_t::detector_type::view_type det_view,
    typename propagator_t::stepper_type::magnetic_field_type field_view,
    vecmem::data::vector_view<
        free_track_parameters<typename propagator_t::algebra_type>>
        tracks_view,
    vecmem::data::jagged_vector_view<
        typename propagator_t::navigator_type::intersection_type>
        nav_cache_view,
    const propagate_option opt) {

    using detector_device_t =
        detector<typename propagator_t::detector_type::metadata,
                 device_container_types>;
    using algebra_t = typename detector_device_t::algebra_type;
    using propagator_device_t =
        propagator<typename propagator_t::stepper_type,
                   navigator<detector_device_t>,
                   typename propagator_t::actor_chain_type>;
    using intersection_t = typename propagator_device_t::intersection_type;

    detector_device_t det(det_view);
    vecmem::device_vector<free_track_parameters<algebra_t>> tracks(tracks_view);
    vecmem::jagged_device_vector<intersection_t> nav_cache(nav_cache_view);

    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= nav_cache.size()) {
        return;
    }

    // Create propagator
    propagator_device_t p{cfg};

    typename parameter_transporter<algebra_t>::state transporter_state{};
    typename pointwise_material_interactor<algebra_t>::state interactor_state{};
    typename parameter_resetter<algebra_t>::state resetter_state{};

    // Create the actor states
    auto actor_states =
        tie(transporter_state, interactor_state, resetter_state);

    // Create the propagator state
    typename propagator_device_t::state p_state(tracks.at(gid), field_view, det,
                                                nav_cache.at(gid));

    // Run propagation
    if (opt == propagate_option::e_unsync) {
        p.propagate(p_state, actor_states);
    } else if (opt == propagate_option::e_sync) {
        p.propagate_sync(p_state, actor_states);
    }
}

template <typename propagator_t>
void run_propagation_kernel(
    const propagation::config& cfg,
    typename propagator_t::detector_type::view_type det_view,
    typename propagator_t::stepper_type::magnetic_field_type field_view,
    vecmem::data::vector_view<
        free_track_parameters<typename propagator_t::algebra_type>>
        tracks_view,
    vecmem::data::jagged_vector_view<
        typename propagator_t::navigator_type::intersection_type>&
        candidates_data,
    const propagate_option opt) {

    constexpr int thread_dim = 256;
    int block_dim =
        static_cast<int>(candidates_data.size() + thread_dim - 1) / thread_dim;

    // run the test kernel
    propagator_benchmark_kernel<propagator_t><<<block_dim, thread_dim>>>(
        cfg, det_view, field_view, tracks_view, candidates_data, opt);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

/// Macro declaring the template instantiations for the different detector types
#define DECLARE_PROPAGATION_BENCHMARK(METADATA, FIELD)                  \
                                                                        \
    template void                                                       \
    run_propagation_kernel<cuda_propagator_type<METADATA, FIELD>>(      \
        const propagation::config&, detector<METADATA>::view_type,      \
        covfie::field_view<FIELD>,                                      \
        vecmem::data::vector_view<                                      \
            free_track_parameters<detector<METADATA>::algebra_type>>,   \
        vecmem::data::jagged_vector_view<                               \
            cuda_propagator_type<METADATA, FIELD>::intersection_type>&, \
        const propagate_option);

DECLARE_PROPAGATION_BENCHMARK(default_metadata, bfield::const_bknd_t)
DECLARE_PROPAGATION_BENCHMARK(toy_metadata, bfield::const_bknd_t)

}  // namespace detray
